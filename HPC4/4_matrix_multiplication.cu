#include "hip/hip_runtime.h"
%%cu
#include <iostream>
using namespace std;


// CUDA code to multiply matrices
__global__ void multiply(int* A, int* B, int* C, int size) {
    // Uses thread indices and block indices to compute each element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}


void initialize(int* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = rand() % 10;
    }
}


void print(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            cout << matrix[row * size + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}


int main() {
    int* A, * B, * C;

    int N = 2;
    int blockSize =  16;

    int matrixSize = N * N;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = new int[matrixSize];
    B = new int[matrixSize];
    C = new int[matrixSize];

    initialize(A, N);
    initialize(B, N);
    cout << "Matrix A: \n";
    print(A, N);

    cout << "Matrix B: \n";
    print(B, N);

    
    int* X, * Y, * Z;
    // Allocate space
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, matrixBytes);
    hipMalloc(&Z, matrixBytes);

    // Copy values from A to X
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);
    
    // Copy values from A to X and B to Y
    hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 2;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = N / THREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // Launch kernel
    multiply<<<blocks, threads>>>(X, Y, Z, N);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    cout << "Multiplication of matrix A and B: \n";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}



// This CUDA C++ code demonstrates matrix multiplication using GPU parallelism. Let's break down the code:

// 1. **Kernel Function (`multiply`)**:
//    - This kernel function is responsible for computing the product of two matrices `A` and `B`.
//    - It takes pointers to the input matrices `A` and `B`, as well as the output matrix `C`, and the size of the matrices (`size`).
//    - Each thread computes one element of the output matrix `C`.
//    - The thread indices (`row` and `col`) are computed using block and thread indices, and each thread iterates over the corresponding row of matrix `A` and column of matrix `B` to calculate the dot product.

// 2. **Helper Functions (`initialize` and `print`)**:
//    - `initialize`: This function initializes a matrix with random values between 0 and 9. It takes a pointer to the matrix and its size as arguments.
//    - `print`: This function prints the elements of a matrix. It takes a pointer to the matrix and its size as arguments.

// 3. **Main Function**:
//    - The main function initializes matrices `A` and `B`, prints them, and then performs matrix multiplication using CUDA.
//    - Matrices `A` and `B` are initialized with random values using the `initialize` function and printed using the `print` function.
//    - Device memory (`X`, `Y`, and `Z`) is allocated using `hipMalloc` for matrices `A`, `B`, and `C` respectively.
//    - The values of matrices `A` and `B` are copied from host to device memory using `hipMemcpy`.
//    - The number of threads per block (`THREADS`) is set to 2, and the number of blocks per grid (`BLOCKS`) is calculated based on the size of the matrices.
//    - The kernel function `multiply` is launched with the specified number of blocks and threads per block.
//    - The result matrix `C` is copied back from device to host memory using `hipMemcpy` and printed.
//    - Finally, memory allocated on the device is freed using `hipFree`.

// 4. **Output**:
//    - The code prints the matrices `A` and `B` before multiplication, and the result matrix `C` after multiplication.

// 5. **Memory Management**:
//    - Memory allocated on the device (`X`, `Y`, and `Z`) is freed at the end of the main function to release GPU resources.

// Overall, this code demonstrates how to leverage GPU parallelism using CUDA to perform matrix multiplication, which can significantly accelerate computation for large matrices compared to sequential CPU-based computation.